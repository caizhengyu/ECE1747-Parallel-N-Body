#include "main.cu"
#include <stdio.h>
// #include <sys/time.h>
#include <stdlib.h>
#include <time.h>
#include <chrono>
// #include "hrtime.h"
using namespace std;

int step;
double steps, endtime;


int main(int argc, char** argv){
    char oFlag = 0;
    if (argc > 2)
        oFlag = 1;

    SetStartTime(0);
    timestep = 0.25;
    LoadFile(argv[1]);

    SetOutToFile("test.txt", 10);

    initialize();

    endtime = 30000;
    steps = endtime/timestep;

    double elapsed = 0;

    auto t_start = chrono::high_resolution_clock::now();
    // the work...

    for(step = 0; step < steps; step++){
        if (oFlag)
            Output();
        Step();
    }
    hipDeviceSynchronize();
    auto t_end = chrono::high_resolution_clock::now();
    elapsed = (t_end - t_start).count();

    fprintf(stdout, "\n==============================\nCode took: %f seconds\n", elapsed / 1000000000.);

    fclose(stdout);
}
