#include "hip/hip_runtime.h"
//This is mainly based on the sequence given by https://conference.sdo.esoc.esa.int/proceedings/sdc7/paper/14/SDC7-paper14.pdf
//The timestep calculation is based on Gravitational N-Body Simulations, SVERRE J. AARSETH

#include <iostream>
#include <map>
#include <vector>
#include <math.h>
#include <fstream>
#include <exception>
#include <sstream>
#include <thrust/copy.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>


using namespace std;

__constant__ double G0=6.6743015e-11, PI0=3.1415926535898;
const double G=6.6743015e-11, PI=3.1415926535898;
thrust::host_vector <double> add_init_position, add_init_velocity, body_values(7, 0.), output_vec, r_rel, v_rel, acting_values(7, 0.), body_final,
					r_f,v_f,accel_output,a_0,a_0_dot,a_p,a_p_dot,
					a_i,a_i_dot,a_j,a_j_dot,r_i,v_i,r_j,v_j,a_ij,a_dot_ij,a_t_dot_ij,a_d_dot_ij,a_d_dot,a_t_dot,
					burn_vector, burn_vector_next, burn_ori, burn_ori_rate;
thrust::host_vector <string> load_results;
double add_mass, r_dot_v_relative, abs_r_rel, a_comp, a_dot_comp_1, a_dot_comp_2, abs_v_rel, comp_a, comp_b, comp_c, body_timestep, abs_a_t_dot, abs_a_d_dot, abs_a_dot, abs_a;
string line, add_id, body_name;
double body_id, acting_id, burn_body, burn_id;
int num_bodies = 0;
int i, itts, time_counter;
int burn_count = 0;
// thrust::pair<string, thrust::host_vector<double> > burn_values, burn_values_next;
thrust::host_vector<double> burn_values, burn_values_next;

map<string,thrust::host_vector<double> > bodies, bodies_next;
map<string, thrust::pair<string, thrust::host_vector<double> > > burns;

thrust::host_vector<string> body_names;

thrust::host_vector<double> burn_data;
thrust::host_vector<double> body_data;
thrust::host_vector<double> body_data_next;

// thrust::device_vector<double> body_ddata;

double accuracy, timestep, time0, next_timestep;
int output_rate;
string info, debug_info;
void SetAccuracy(int new_accuracy){
	accuracy=new_accuracy;
}
void SetStartTime(int start_time){
	time0=start_time;
}
void SetOutToFile(const char * file_name, int rate){
	freopen(file_name,"w",stdout);
	output_rate = rate;
}
void AddBody (string id, double mass, thrust::host_vector<double> init_position, thrust::host_vector<double> init_velocity){
    bodies[id].clear();//so adding bodies with the same id twice doesn't break it
	bodies[id].push_back(mass);

	for(i=0;i<init_position.size();i++){
		bodies[id].push_back(init_position[i]);
	}

	for(i=0;i<init_velocity.size();i++){
		bodies[id].push_back(init_velocity[i]);
	}
}

void AddBody2(string id, double mass, thrust::host_vector<double> init_position, thrust::host_vector<double> init_velocity){
	body_names.push_back(id);
	body_data.push_back(mass);

	for(i=0;i<init_position.size();i++){
		body_data.push_back(init_position[i]);
	}

	for(i=0;i<init_velocity.size();i++){
		body_data.push_back(init_velocity[i]);
	}
}

void AddBurn (double burn_id, double body_id, double start_time, double end_time, double acceleration, thrust::host_vector<double> orientation, thrust::host_vector<double> orientation_rate){
	//Gives burn_id:(body_id:[start,end,accel,orie_x,orie_y,orie_z,orie_ra_x...])
	// burn_vector.clear();
	burn_data.push_back(burn_id);
	burn_data.push_back(body_id);
	burn_data.push_back(start_time);
	burn_data.push_back(end_time);
	burn_data.push_back(acceleration);
	for(i=0;i<3;i++){
		// burn_vector.push_back(orientation[i]);
		burn_data.push_back(orientation[i]);
	}
	for(i=0;i<3;i++){
		// burn_vector.push_back(orientation_rate[i]);
		burn_data.push_back(orientation_rate[i]);
	}
	// burns[burn_id] = thrust::make_pair(body_id,burn_vector);
}


void LoadFile(string filename){
	//Format is #body,id,mass,position_x,position_y,position_z,velocity_x,velocity_y,velocity_z
	body_data.clear();
	ifstream file(filename);
	if (file.is_open()) {
		string line;
		int j = 0;
		while (getline(file, line)) {
			if(line.compare(0,1,"#")==0){
				if(line.compare(1, 4,"Body")==0){
					load_results.clear();
					stringstream s_stream(line);
					while(s_stream.good()) {
					    string substr;
					    getline(s_stream, substr, ',');
					    load_results.push_back(substr);
					}
					add_init_position.clear();
					add_init_velocity.clear();

					add_id = load_results[1];
					add_mass = stold(load_results[2]);

					for(i=0;i<3;i++){
						add_init_position.push_back(stold(load_results[3+i]));
						add_init_velocity.push_back(stold(load_results[6+i]));
					}
					// AddBody(add_id, add_mass, add_init_position, add_init_velocity);
					AddBody2(add_id, add_mass, add_init_position, add_init_velocity);
					num_bodies++;
				}
				if(line.compare(1, 4,"Burn")==0){
					load_results.clear();
					stringstream s_stream(line);
					while(s_stream.good()) {
					    string substr;
					    getline(s_stream, substr, ',');
					    load_results.push_back(substr);
					}
					burn_ori.clear();burn_ori_rate.clear();
					for(i=0;i<3;i++){
						burn_ori.push_back(stold(load_results[5+i]));
						burn_ori_rate.push_back(stold(load_results[8+i]));
					}
					// burn_id = load_results[1];
					cout<<load_results[1];
					AddBurn(burn_count,num_bodies-1,stold(load_results[2]),stold(load_results[3]),stold(load_results[4]),burn_ori,burn_ori_rate);
					burn_count++;
				}
				if(line.compare(1, 8,"Addition")==0){
					cout<<"This functionality has not yet been added";
					throw exception();
				}
				++j;
			}
		}
		body_data_next = body_data;
		file.close();
	}
}


#if __CUDA_ARCH__ < 600
__device__ double atomicAdd1(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}
#endif


__device__ void CalcAAndDotD(double* output_vec, double body_id, double* r, double* v, double* body_data, 
	double* burn_data, size_t bodySize, size_t burnSize, double time0){
	// vector<double> a, a_dot, acting_values, r_rel, v_rel;
	double acting_values[7], r_rel[3], v_rel[3], a[3], a_dot[3];
	double acting_id, r_dot_v_relative, abs_r_rel, a_comp, a_dot_comp_1, a_dot_comp_2;
	int i;

	for(i = 0;i < 3;i++){
		a[i] = 0;
		a_dot[i] = 0;
	}
	acting_id = threadIdx.x;
	int q = 7 * acting_id;
	if(acting_id != body_id){
		// acting_values = body_itterator.second;
		for (i = 0; i < 7; i++)
			acting_values[i] = body_data[q+i];
		for(i=0;i<3;i++){
			r_rel[i] = r[i] - acting_values[1+i];
		}
		for(i=0;i<3;i++){
			v_rel[i] = v[i] - acting_values[4+i];
		}
		for(i=0;i<3;i++){
			r_dot_v_relative = r_rel[i] * v_rel[i];
		}

		abs_r_rel = sqrt(pow(r_rel[0],2)+pow(r_rel[1],2)+pow(r_rel[2],2));

		a_comp = -G0*acting_values[0]/pow(abs_r_rel,3);
		a_dot_comp_1 = 3*G0*acting_values[0]*r_dot_v_relative/pow(abs_r_rel,5);
		a_dot_comp_2 = -G0*acting_values[0]/pow(abs_r_rel,3);

		for(i=0;i<3;i++){
			a[i] += a_comp*r_rel[i];
			a_dot[i] += a_dot_comp_1*r_rel[i]+a_dot_comp_2*v_rel[i];
		}
	}

	for(i=0;i<3;i++){
		// output_vec[i] = a[i];
		if (a[i])
			atomicAdd1(&output_vec[i], a[i]);
	}
	for(i=0;i<3;i++){
		// output_vec[i+3] = a_dot[i];
		if (a_dot[i])
			atomicAdd1(&output_vec[i + 3], a_dot[i]);
	}
	__syncthreads();
	if (threadIdx.x)
		return;
	// master thread shall work on burn, if any
	double burn_body;
	double burn_values[11], burn_vector[10];

	for (int q = 0; q < burnSize; q += 11){
		for (i = 0; i < 11; i++){
			// burn_values.push_back(burn_data[q + i]);
			burn_values[i] = burn_data[q + i];
		}
		// burn_values = burn_itt.second;
		burn_body = burn_values[1];
		if(burn_body == body_id){
			for (i = 1; i < 11; i++){
				// burn_vector.push_back(burn_values[i]);
				burn_vector[i - 1] = burn_values[i];
			}
			// burn_vector = burn_values.second;
			if(time0>burn_vector[0]&&time0<burn_vector[1]){
				for(i=0;i<3;i++){
					output_vec[i] = burn_vector[2]*burn_vector[3+i];
				}
			}
		}
	}
}


__global__ void perform(double* body_data0, double* burn_data, size_t bodySize, size_t burnSize, double time0, double timestep, 
	double* body_data_next) {
	int i, q;
	// create shared array for every block instead of using global data
	__shared__ double body_data[100];
	for (i =0; i < 7; i++)
		body_data[threadIdx.x * 7 + i] = body_data0[threadIdx.x * 7 + i];
	__syncthreads();

	double r_0[3], v_0[3];
	double body_values[7], body_id;
	q = blockIdx.x * 7;
	for (i = 0; i < 7; i++)
		body_values[i] = body_data[q+i];
	body_id = q / 7;

	for(i=0;i<3;i++){
		r_0[i] = body_values[1+i];
		v_0[i] = body_values[4+i];
	}

	__shared__ double accel_output[6];
	if (!threadIdx.x)
		for (i = 0; i < 6; i++)
			accel_output[i] = 0;
	__syncthreads();
	CalcAAndDotD(accel_output, body_id, r_0, v_0, body_data, burn_data, bodySize, burnSize, time0);
	// barrier not really needed here but whatever
	__syncthreads();
	__shared__ double r_p[3], v_p[3];
	double a_0[3], a_0_dot[3];
	if (!threadIdx.x){
		for(i=0;i<3;i++){
			a_0[i] = accel_output[i];
			a_0_dot[i] = accel_output[3+i];
		}
		
		for(i=0;i<3;i++){
			r_p[i] = r_0[i]+v_0[i]*timestep+0.5*a_0[i]*pow(timestep,2)+(1/6)*a_0_dot[i]*pow(timestep,3);
			v_p[i] = v_0[i]+a_0[i]*timestep+0.5*a_0_dot[i]*pow(timestep,2);
		}
	}	

	double a_p[3], a_p_dot[3];
	for(int itts=0;itts<2;itts++){
		__syncthreads();
		CalcAAndDotD(accel_output, body_id, r_p, v_p, body_data, burn_data, bodySize, burnSize, time0);
		// barrier not really needed here but whatever
		__syncthreads();
		if (!threadIdx.x){
			for(i=0;i<3;i++){
				a_p[i] = accel_output[i];
				a_p_dot[i] = accel_output[3+i];
			}

			for(i=0;i<3;i++){
				v_p[i] = v_0[i]+0.5*(a_0[i]+a_p[i])*timestep+(1/12)*(a_0_dot[i]-a_p_dot[i]*pow(timestep,2));
				r_p[i] = r_0[i]+0.5*(v_p[i]+v_0[i])*timestep+(1/12)*(a_0[i]-a_p[i])*pow(timestep,2);
			}
		}
	}
	// only master thread is premitted to write to global data
	if (threadIdx.x) return;
	double body_final[7];
	body_final[0] = body_values[0];

	for(i=0;i<3;i++){
		body_final[i + 1] = r_p[i];
	}
	for(i=0;i<3;i++){
		body_final[i + 4] = v_p[i];
	}
	// begin
	for (i = 0; i < 7; i++){
		body_data_next[q+i] = body_final[i];
	}
}


thrust::device_vector<double> body_ddata, burn_ddata, body_ddata_next;
double* body_ddata_ptr, *burn_ddata_ptr, *body_ddata_next_ptr;
void initialize(){
	body_ddata = body_data;
	body_ddata_ptr = thrust::raw_pointer_cast(body_ddata.data());
	burn_ddata = burn_data;
	burn_ddata_ptr = thrust::raw_pointer_cast(burn_ddata.data());
	body_ddata_next = body_data;
	body_ddata_next_ptr = thrust::raw_pointer_cast(body_ddata_next.data());
}


void swapPtr(double **r, double **s)
{
    double *pSwap = *r;
    *r = *s;
    *s = pSwap;
}


void Step() {
	next_timestep = 9999999999;
	// begin
	perform<<<num_bodies, num_bodies>>>(body_ddata_ptr, burn_ddata_ptr, body_data.size(), burn_data.size(), time0, timestep, 
		body_ddata_next_ptr);
	// body_ddata = body_ddata_next;
	swapPtr(&body_ddata_ptr, &body_ddata_next_ptr);
	time0 = time0 + timestep;
}


void Output(){
	if(time_counter==output_rate){
		cout<<"#"+to_string(time0)+"\n";
		// begin
		for (int q = 0; q < body_data.size(); q+=7){
			for (i = 0; i < 7; i++)
				// copy from gpu to cpu
				body_values[i] = body_ddata[q+i];
			body_id = q / 7;
			body_name = body_names[body_id];
			cout<<body_name;
			for(i = 0; i < body_values.size(); i++){
				cout << "," + to_string(body_values[i]);
			}
			cout<<"\n";
		}
		time_counter=0;
	}
	else{
		time_counter++;
	}
}
